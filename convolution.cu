#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 4
#define TPB 4
#define INPUT_SIZE 12
#define MAX_MASK_WIDTH 5
__constant__ float M[MAX_MASK_WIDTH];

__global__ void convolution_shared_memory(float *N, float *P){
	
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	__shared__ float N_ds[TILE_SIZE];

	N_ds[threadIdx.x]=N[i];

	__syncthreads();

	int this_title_start_point = blockIdx.x*blockDim.x;
	int next_tile_start_point = (blockIdx.x+1)*blockDim.x;
	int N_start_point = i-(MAX_MASK_WIDTH/2);
	float Pvalue = 0;


	for(int j =0; j < MAX_MASK_WIDTH; j++){

		int N_index = N_start_point+j;

		if(N_index >=0 && N_index < INPUT_SIZE){
			if((N_index>= this_title_start_point) && (N_index<next_tile_start_point)){
				Pvalue+=N_ds[threadIdx.x+j-(MAX_MASK_WIDTH/2)]*M[j];
			}
			else{
				Pvalue+=N[N_index]*M[j];
			}
		}
	}

	P[i]=Pvalue;	
}

int main(){

	//device input and output
	float *d_N = 0;
	float *d_P = 0;

	hipMalloc(&d_N,INPUT_SIZE*sizeof(float));
	hipMalloc(&d_P,INPUT_SIZE*sizeof(float));


	//host input and output
	float *h_N = (float*)malloc(INPUT_SIZE*sizeof(float));
	float *h_P = (float*)malloc(INPUT_SIZE*sizeof(float));
	float *h_M = (float*)malloc(MAX_MASK_WIDTH*sizeof(float));

	//initialize input on host
	for(int i=0;i<INPUT_SIZE;++i){
		h_N[i]=(float)i;
	}

	//transfer input to device
	hipMemcpy(d_N,h_N,INPUT_SIZE*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_P,h_P,INPUT_SIZE*sizeof(float),hipMemcpyHostToDevice);

	//initialize mask on host
	for(int j=0;j<MAX_MASK_WIDTH;++j){
		h_M[j]=(float)j;
	}

	//transfer mask to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(M),h_M,MAX_MASK_WIDTH*sizeof(float));


	//call convolution kernel
	convolution_shared_memory<<<(INPUT_SIZE+TPB-1)/TPB,TPB >>>(d_N,d_P);

	//retrieve result from device
	hipMemcpy(h_P,d_P,INPUT_SIZE*sizeof(float),hipMemcpyDeviceToHost);

	for(int i=0; i<INPUT_SIZE;++i){
		printf("%f\n", h_P[i]);
	}


	hipFree(d_N);
	hipFree(d_P);
	hipFree(M);

	free(h_N);
	free(h_P);
	free(h_M);

	printf("Hello world \n");

}